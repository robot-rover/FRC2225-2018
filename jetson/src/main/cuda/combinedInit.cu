
#include <hip/hip_runtime.h>
extern "C"
__device__ int clamp(int x, int a, int b) {
  return max(a, min(b, x));
}

extern "C"
__global__ void combinedInit(unsigned char *ra, unsigned char *ga, unsigned char *ba, float *val, const int size) {
    int gid = blockIdx.x *blockDim.x + threadIdx.x;
    if (gid < size) {
        // Begin rgbToHsv
        unsigned char r = ra[gid];
        unsigned char g = ga[gid];
        unsigned char b = ba[gid];
        float hue;
        float sat;
        unsigned char mx = max(r, g);
        mx = max(mx, b);
        unsigned char mn = (float) min(r, g);
        mn = min(mn, b);

        float delta = (float) (mx - mn);
        //bv[gid] = mx;

        if(mx == 0) {
            val[gid] = 0.0f;
            return;
        } else {
            sat = delta / (float) mx;

            if(r == mx)
                hue = (float)(g - b) / delta;
            else if( g == mx )
                hue = 2 + (float)( b - r ) / delta;
            else
                hue = 4 + (float)( r - g ) / delta;

            hue = hue * 1.0471976f;

            if (hue < 0)
                hue = hue + 6.2831855;
        }

        // Begin hue

        float sqr = hue - 1.07f;
        sqr = fabs(sqr) - 0.3f;
        sqr = max(sqr, 0.0f);
        sqr = sqrt(sqr);
        float sVal = 0.5f - sat;
        mx = max(0.0f, sVal);
        mx = mx * 100.0f;
        sqr = sqr - mx;
        sqr = 5.0f/sqr;
        val[gid] = clamp(sqr, 0.0f, 255.0f);
    }
}