
#include <hip/hip_runtime.h>
extern "C"
__global__ void interToPlanar(const char *interleaved, char *r, char *g, char *b, const int size) {
    int gid = blockIdx.x *blockDim.x + threadIdx.x;
    if(gid < size) {
        int index = gid * 3;
        r[gid] = interleaved[index+2];
        g[gid] = interleaved[index+1];
        b[gid] = interleaved[index];
    }
}