
#include <hip/hip_runtime.h>
extern "C"
__global__ void scaleThresh(float *src, int *dst, const int scale, const int width, const int height, const float threshold, const int down) {
    int gid  = blockIdx.x *blockDim.x + threadIdx.x;
    int size = width * height;
    int downSqr = scale * scale;
    if(gid < size / downSqr) {
        int gidPerRow = width / scale;
        int y = gid / gidPerRow;
        y = y * scale;
        int yMax = y + scale;
        int x = gid % gidPerRow;
        x = x * scale;
        int xInit = x;
        int xMax = x + scale;
        float val = 0;
        int addr;
        for(; y < yMax; y++) {
            x = xInit;
            for(; x < xMax; x++) {
                addr = y * width + x;
                val = val + src[addr];
            }
        }

        if( val / downSqr <= threshold) {
            dst[gid] = down;
        } else {
            dst[gid] = !down;
        }
    }
}