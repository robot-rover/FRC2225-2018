
#include <hip/hip_runtime.h>
extern "C"
__device__ int clamp(int x, int a, int b){
  return max(a, min(b, x));
}

extern "C"
__device__ int getPixel(int x, int y, int width, int height) {
    int xBound = clamp(x, 0, width - 1);
    int yBound = clamp(y, 0, height - 1);
    return xBound + yBound * width;
}

extern "C"
__global__ void blurMeanBin(int *src, int *dst, const int width, const int height, const int radius) {
    int size = width * height;
    int gid = blockIdx.x *blockDim.x + threadIdx.x;
    if (gid < size) {
        int x = gid % width;
        int xMx = x + radius;
        int xMn = x - radius;
        int y = gid / width;
        int yMx = y + radius;
        y = y - radius;
        float mean = 0;
        for (; y <= yMx; y++) {
            for(x = xMn; x <= xMx; x++) {
                mean = mean + src[getPixel(x, y, width, height)];
            }
        }
        mean = mean / (2*radius + 1) / (2*radius+1);
        if(mean > 0.81) {
            dst[gid] = 1;
        } else {
            dst[gid] = 0;
        }
    }
}